#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include "kernels.h"

#define cudaCheckError() {                                                                       \
  hipError_t e=hipGetLastError();                                                        \
  if(e!=hipSuccess) {                                                                     \
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
      exit(EXIT_FAILURE);                                                                  \
  }                                                                                        \
}

// Check that all values of array (which contains `length` float elements) are
// close to `expectedValue`
bool checkHostArray(float *array, float expectedValue, size_t length){
  float maxError = 0.0f;
  for (int i = 0; i < length; i++)
    maxError = fmax(maxError, fabs(array[i]-expectedValue));
  std::cout << "Max error: " << maxError << std::endl;
  return (maxError < 0.0001f);
}

int main(void)
{
  int N = 1<<20;  //< Number of elements in arrays (1M, you may want to lower this to begin)
  float *d_x;  //< Pointer to the 1D buffer we will manipulate 
 
  // Initialize the x and y arrays on the device
  // use as many threads as possible, 
  // and assume we will process each element of d_x
  // with a different thread
  //@@ Choose some values here, stick to 1D
  int threadsPerBlock = 512;
  int blocksPerGrid = N / threadsPerBlock;

  // Array allocation on device
  //@@ Use hipMalloc to perform the allocation.
  hipMalloc((void**)&d_x, N * sizeof(float));
  cudaCheckError();
 
  // Initialize the x and y arrays on the device
  const float firstValue = 1.f;
  //@@ Call the fill1D kernel to fill d_x with `firstValue`, see kernels.h for the API
  fill1D<<<blocksPerGrid, threadsPerBlock>>>(d_x, firstValue, N);
  // Wait for GPU to finish and check for errors
  hipDeviceSynchronize();
  cudaCheckError();
  
  // Check for errors on device
  float expectedValue = firstValue;
  //@@ Call the check1D kernel to control device memory content, see kernels.h for API
  check1D<<<blocksPerGrid, threadsPerBlock>>>(d_x, expectedValue, N);
  // Wait for GPU to finish and check for errors
  //@@ call CUDA device synchronisation function
  hipDeviceSynchronize();
  cudaCheckError();

  // Copy back the buffer to the host for inspection:
  //@@ Allocate a buffer on the host
  float* h_x = (float*) std::malloc(N * sizeof(float));
  //@@ Copy the buffer content from device to host
  //@@ use hipMemcpy
  hipMemcpy(h_x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError();

  // Check for errors (all values should be close to `firstValue`)
  std::cout << "First control..." << std::endl;
  bool noerror = checkHostArray(h_x, firstValue, N);
  
  // Now increment the array values by some other value
  const float otherValue = 10.f;
  //@@ Call the inc1D kernel to add `otherValue` to all values of our buffer, see kernels.h for API
  inc1D<<<blocksPerGrid, threadsPerBlock>>>(d_x, otherValue, N);
  // Wait for GPU to finish
  //@@ call CUDA device synchronisation function
  hipDeviceSynchronize();
  cudaCheckError();

  // Check for errors on device
  expectedValue = firstValue + otherValue;
  //@@ Call the check1D kernel to control device memory content, see kernels.h for API
  check1D<<<blocksPerGrid, threadsPerBlock>>>(d_x, expectedValue, N);
  // Wait for GPU to finish and check for errors
  //@@ call CUDA device synchronisation function
  hipDeviceSynchronize();
  cudaCheckError();

  // Copy back the buffer to the host for inspection:
  //@@ Copy the buffer content from device to host (reuse previous buffer)
  hipMemcpy(h_x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckError();

  // Check for errors (all values should be close to `firstValue+otherValue`)
  std::cout << "Second control..." << std::endl;
  noerror &= checkHostArray(h_x, firstValue+otherValue, N);

  // Free memory
  //@@ free d_h using CUDA primitives 
  hipFree(d_x);
  cudaCheckError();
  std::free(h_x);

  if (noerror) {
    printf("Test completed successfully.\n");
    return 0;
  } else {
    printf("WARNING there were some errors.\n");
    return 1;
  }
}
